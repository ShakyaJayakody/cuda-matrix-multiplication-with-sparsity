#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define ROW 10
#define COL 10
#define SPARSITY 80
#define BLOCK_SIZE 16

__global__ void matrix_mul(int* a, int* b, int* c, int row, int col) {
    int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (rowIdx < row && colIdx < col) {
        int tmp = 0;
        for (int i = 0; i < col; i++) {
            tmp += a[rowIdx * col + i] * b[i * col + colIdx];
        }
        c[rowIdx * col + colIdx] = tmp;
    }
}

void initilizeMatrix(int* m, int row, int col) {
    float dense_rate = (100.0 - float(SPARSITY)) / 100.0;
    int total_elements = row * col;
    int dense_elements = dense_rate * total_elements;

    //srand(time(0));
    int i = 0;
    while (i < dense_elements) {
        int rnd_row = rand() % row;
        int rnd_col = rand() % col;
        int index = rnd_row * col + rnd_col;

        if (m[index] == 0) {
            m[index] = (rand() % 100) + 1;
            i++;
        }
    }
}

void printMatrix(int* m, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%5d ", m[i * col + j]);
        }
        printf("\n");
    }
}

int main() {
    int* h_a, * h_b, * h_c;
    int* d_a, * d_b, * d_c;

    h_a = (int*)malloc(sizeof(int) * ROW * COL);
    h_b = (int*)malloc(sizeof(int) * COL * ROW);
    h_c = (int*)malloc(sizeof(int) * ROW * ROW);

    memset(h_a, 0, sizeof(int) * ROW * COL);
    memset(h_b, 0, sizeof(int) * COL * ROW);
    memset(h_c, 0, sizeof(int) * ROW * ROW);

    initilizeMatrix(h_a, ROW, COL);
    initilizeMatrix(h_b, COL, ROW);

    printf("Matrix A:\n");
    printMatrix(h_a, ROW, COL);
    printf("\nMatrix B:\n");
    printMatrix(h_b, COL, ROW);

    hipMalloc((void**)&d_a, sizeof(int) * ROW * COL);
    hipMalloc((void**)&d_b, sizeof(int) * COL * ROW);
    hipMalloc((void**)&d_c, sizeof(int) * ROW * ROW);

    hipMemcpy(d_a, h_a, sizeof(int) * ROW * COL, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * COL * ROW, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((ROW + dimBlock.x - 1) / dimBlock.x, (ROW + dimBlock.y - 1) / dimBlock.y);

    matrix_mul << <dimGrid, dimBlock >> > (d_a, d_b, d_c, ROW, ROW);

    hipMemcpy(h_c, d_c, sizeof(int) * ROW * ROW, hipMemcpyDeviceToHost);

    printf("\nMatrix C:\n");
    printMatrix(h_c, ROW, ROW);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
